#include "hip/hip_runtime.h"
/*Copyright 2025 He Jia <mofhejia@163.com>. All Rights Reserved.

Licensed under the Apache License Version 2.0 with LLVM Exceptions
(the "License"); you may not use this file except in compliance with
the License. You may obtain a copy of the License at

    https://llvm.org/LICENSE.txt

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#ifndef UCX_AM_CONTEXT_TEST_CUDA_HELPER_CUH_
#define UCX_AM_CONTEXT_TEST_CUDA_HELPER_CUH_

#include "ucx_context/ucx_am_context/ucx_am_context_test_helper.h"

#include "ucx_context/ucx_context_def.h"

#include <cstdio>
#include <hip/hip_runtime.h>

// CUDA kernel to divide each element of a float array by 2.0f
__global__ void divide_by_two_kernel_cuh(float* data, size_t n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    data[idx] /= 2.0f;
  }
}

/**
 * @brief Processes received data on the CUDA device.
 *
 * This function takes a pointer to ucx_am_data structure. It assumes that
 * recvData->data points to CUDA device memory and recvData->data_type is
 * ucx_memory_type::CUDA. The function divides each float element in
 * recvData->data by 2.0f.
 *
 * @param recvData Pointer to the ucx_am_data structure containing the data to
 * be processed. The data within must be float type and reside in CUDA device
 * memory.
 */
__attribute__((visibility("default"))) void processRecvDataCuda(
  ucx_am_data_t& recvData) {
  if (recvData.data == nullptr || recvData.data_length == 0) {
    fprintf(stderr, "processRecvDataCuda: Received null or empty data.\n");
    return;
  }

  // This function assumes that recvData->data points to CUDA device memory
  // and recvData->data_type is ucx_memory_type::CUDA.
  float* d_data = static_cast<float*>(recvData.data);
  size_t num_elements = recvData.data_length / sizeof(float);

  if (num_elements == 0) {
    return;  // No elements to process
  }

  // Configure kernel launch parameters
  int threads_per_block = 256;  // Common default, can be tuned
  int blocks_per_grid =
    (num_elements + threads_per_block - 1) / threads_per_block;

  // Launch the kernel
  divide_by_two_kernel_cuh<<<blocks_per_grid, threads_per_block>>>(
    d_data, num_elements);

  hipError_t lastError = hipGetLastError();
  if (lastError != hipSuccess) {
    fprintf(
      stderr, "CUDA error in processRecvDataCuda kernel launch: %s\n",
      hipGetErrorString(lastError));
  }
  hipDeviceSynchronize();
}

#endif  // UCX_AM_CONTEXT_TEST_CUDA_HELPER_CUH_
